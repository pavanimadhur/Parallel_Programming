#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
extern void init(int *r, int *c, long long int *a1, long long int *a2, long long int *a3, long long int *a4, long long int *a5);
extern void print2(long long int *mat, long long int r, long long int c, char* str)
{
	long long int i, j;
	FILE *fptr;

	fptr = fopen(str, "a");
	//printf("Storing in file:%s\n",str);
	fprintf(fptr, "ARRAY\n");
	if (fptr == NULL)
	{
		printf("Error!");
		exit(1);
	}
	for (i = 0; i<r; i++)
	{
		for (j = 0; j<c; j++)
		{
			fprintf(fptr, "%lld ", mat[i*c + j]);
		}fprintf(fptr, "\n");
	}fprintf(fptr, "\n");
	fclose(fptr);
}

typedef struct ll
{
	char c;
	long long int *mat;
	long long int rv;
	long long int cv;
	struct ll *next;
}node;

extern __global__ void MatrixMul(long long int* A, long long int * B, long long int * C,
	long long int numARows, long long int numAColumns, long long  int numBColumns)
{
	__shared__ long long  int sA[32][32];   // Tile size of 32x32 
	__shared__  long long int sB[32][32];

	long long int Row = blockDim.y*blockIdx.y + threadIdx.y;
	long long int Col = blockDim.x*blockIdx.x + threadIdx.x;
	long long int Cvalue = 0;
	sA[threadIdx.y][threadIdx.x] = 0;
	sB[threadIdx.y][threadIdx.x] = 0;

	for (int k = 0; k < (((numAColumns - 1) / 32) + 1); k++)
	{
		if ((Row < numARows) && (threadIdx.x + (k * 32)) < numAColumns)
		{
			sA[threadIdx.y][threadIdx.x] = A[(Row*numAColumns) + threadIdx.x + (k * 32)];
		}
		else
		{
			sA[threadIdx.y][threadIdx.x] = 0.0;
		}
		if (Col < numBColumns && (threadIdx.y + k * 32) < numAColumns)
		{
			sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k * 32)*numBColumns + Col];
		}
		else
		{
			sB[threadIdx.y][threadIdx.x] = 0.0;
		}
		__syncthreads();

		for (int j = 0; j < 32; ++j)
		{
			Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
		}
	}
	if (Row < numARows && Col < numBColumns)
	{
		C[Row*numBColumns + Col] = Cvalue;
	}
}


extern void call(int *r, int *c,char *str)
{
	//const int w=10;
	long long int i, j, f = 0;
	long long int zero = 0;
	char stk[256];

	long long int *arr1 = (long long int *)malloc(r[0] * c[0] * sizeof(long long int));
	long long int *arr2 = (long long int *)malloc(r[1] * c[1] * sizeof(long long int));
	long long int *arr3 = (long long int *)malloc(r[2] * c[2] * sizeof(long long int));
	long long int *arr4 = (long long int *)malloc(r[3] * c[3] * sizeof(long long int));
	long long int *arr5 = (long long int *)malloc(r[4] * c[4] * sizeof(long long int));
	long long int *count1 = (long long int *)malloc(r[0] * c[0] * sizeof(long long int));
	for (i = 0; i<r[0]; i++)
	{
		for (j = 0; j<c[0]; j++)
		{
			count1[i*c[0] + j] = 0;
		}
	}
	FILE *fptr;

	fptr = fopen("output.txt", "a");
	init(r, c, arr1, arr2, arr3, arr4, arr5);
	/*
	print2(arr1, r[0], c[0], "inp_parallel.txt");
	print2(arr2, r[1], c[1], "inp_parallel.txt");
	print2(arr3, r[2], c[2], "inp_parallel.txt");
	print2(arr4, r[3], c[3], "inp_parallel.txt");
	print2(arr5, r[4], c[4], "inp_parallel.txt");*/
	node *head = NULL, *temp, *temp2 = NULL, *temp3 = NULL;
	head = (node *)malloc(sizeof(node));
	head->c = 'A';
	head->mat = arr1;
	head->rv = r[0];
	head->cv = c[0];
	temp = head;
	temp->next = (node *)malloc(sizeof(node));
	temp = temp->next;
	temp->c = 'B';
	temp->rv = r[1];
	temp->cv = c[1];
	temp->mat = arr2;
	temp->next = (node *)malloc(sizeof(node));
	temp = temp->next;
	temp->c = 'C';
	temp->rv = r[2];
	temp->cv = c[2];
	temp->mat = arr3;
	temp->next = (node *)malloc(sizeof(node));
	temp = temp->next;
	temp->c = 'D';
	temp->rv = r[3];
	temp->cv = c[3];
	temp->mat = arr4;
	temp->next = NULL;
	temp->next = (node *)malloc(sizeof(node));
	temp = temp->next;
	temp->c = 'E';
	temp->rv = r[4];
	temp->cv = c[4];
	temp->mat = arr5;
	temp->next = NULL;

	
	//fgets(str, sizeof(str), stdin);
	i = 0;
	j = 0;

	clock_t t;
	t = clock();
	while (i<strlen(str))
	{
		while (str[i] != ')' && i<strlen(str))
		{
			if (str[i] == '(')
			{
				i++;
			}
			else
			{
				stk[j] = str[i];
				j++;
				i++;
			}
		}
		if (str[i] == ')')
		{
			char x = stk[--j];
			char y = stk[--j];
			temp2 = head;
			temp = head;
			while (temp->next != NULL)
			{
				if (temp->c == y)
				{
					temp2 = temp;
					break;
				}
				temp = temp->next;
			}
			temp = head;
			while (temp->next != NULL)
			{
				if (temp->c == x)
				{
					temp3 = temp;
					break;
				}
				temp = temp->next;
			}

			long long int *mat1_d, *mat2_d, *result_d, *count_d, *rv_d, *cv_d, *cv2_d;
			hipMalloc((void**)&mat1_d, temp2->rv * temp2->cv * sizeof(long long int));
			hipMalloc((void**)&mat2_d, temp->rv * temp->cv * sizeof(long long int));
			hipMalloc((void**)&result_d, temp2->rv * temp->cv * sizeof(long long int));
			hipMalloc((void**)&count_d, temp2->rv * temp->cv * sizeof(long long int));

			hipMalloc((void **)&rv_d, sizeof(long long int));
			hipMalloc((void **)&cv_d, sizeof(long long int));
			hipMalloc((void **)&cv2_d, sizeof(long long int));

			hipMemcpy(mat1_d, temp2->mat, temp2->rv * temp2->cv * sizeof(long long int), hipMemcpyHostToDevice);
			hipMemcpy(mat2_d, temp->mat, temp->rv * temp->cv * sizeof(long long int), hipMemcpyHostToDevice);

			hipMemcpy(rv_d, &temp2->rv, sizeof(long long int), hipMemcpyHostToDevice);
			hipMemcpy(cv_d, &temp2->cv, sizeof(long long int), hipMemcpyHostToDevice);
			hipMemcpy(cv2_d, &temp->cv, sizeof(long long int), hipMemcpyHostToDevice);

			dim3 dimBlock(32, 32, 1);
			dim3 dimGrid((temp->cv / 32) + 1, (temp2->rv / 32) + 1, 1);



			MatrixMul << <dimGrid, dimBlock >> >(mat1_d, mat2_d, result_d, temp2->rv, temp2->cv, temp->cv);
			printf("\n");
			temp2->mat = (long long int *)realloc(temp2->mat, temp2->rv * temp->cv * sizeof(long long int));
			count1 = (long long int *)realloc(count1, temp2->rv * temp->cv * sizeof(long long int));
			hipMemcpy(temp2->mat, result_d, temp2->rv * temp->cv * sizeof(long long int), hipMemcpyDeviceToHost);
			hipMemcpy(count1, count_d, temp2->rv * temp->cv * sizeof(long long int), hipMemcpyDeviceToHost);
			for (f = 0; f<temp2->rv * temp->cv; f++)
			{
				zero += count1[f];
			}
			//final+=zero;
			temp2->cv = temp->cv;
			stk[++j] = temp2->c;
			i++;
			//print2(temp2->mat, temp2->rv, temp2->cv, "prog_parallel.txt");
			hipError_t err = hipFree(mat1_d);
			//printf("Free error: %s\n", hipGetErrorString(err));
			hipFree(mat2_d);
			hipFree(result_d);
			hipFree(count_d);
			hipFree(rv_d);
			hipFree(cv_d);
			hipFree(cv2_d);
		}
	}
	//print2(temp2->mat,temp2->rv, temp2->cv, "outp_parallel.txt");
	t = clock() - t;
	double time_taken = ((double)t) / CLOCKS_PER_SEC; // in seconds
	fprintf(fptr, "Parallel Multiplication:");
	printf("Parallel Multiplication:");
	fprintf(fptr,"fun() took %f seconds to execute \n", time_taken);
	printf("fun() took %f seconds to execute \n", time_taken);
	//fprintf(fptr, "-------------------------------\n");
	//printf("no. of multiplications reduced due to zeroes:%lld\n", zero);
	
	//system("pause");
}