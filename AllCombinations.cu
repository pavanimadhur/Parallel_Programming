#include "hip/hip_runtime.h"


#include <stdio.h>
#include<stdlib.h>
#include <string.h>
#include <time.h>
extern char*  AllCombi(int n, int *rc, int *cc)
{
	long long int i, mul1[10], mul2[10], mul3[20];
	long long int r[5], c[5],minimum;
	char *str;
	FILE *fptr;

	fptr = fopen("output.txt", "a");
	//printf("working:\n");
	for (i = 0; i < n; i++)
	{
		r[i] = rc[i];
		c[i] = cc[i];
	}
	if (n == 3)
	{
		fprintf(fptr, "A:%lldx%lld   B:%lldx%lld   C:%lldx%lld", r[0], c[0], r[1], c[1], r[2], c[2]);
		fprintf(fptr, "\n order       No.of multiplications    ");
		mul1[0] = (r[1] * c[1] * c[2]) + (r[0] * c[0] * c[2]);
		fprintf(fptr, "\n(A(BC))\t%lld", mul1[0]);
		fprintf(fptr, "\n");
		mul1[1] = (r[0] * c[0] * c[1]) + (r[0] * c[1] * c[2]);
		fprintf(fptr, "\n((AB)C)\t%lld", mul1[1]);
		fprintf(fptr, "\n");
		if (mul1[0]<mul1[1])
		{
			str="(A(BC))";
		}
		else
		{
			str="((AB)C)";
		}
		
	}
	else if (n == 4)
	{
		fprintf(fptr, "A:%lldx%lld   B:%lldx%lld   C:%lldx%lld   D:%lldx%lld", r[0], c[0], r[1], c[1], r[2], c[2], r[3], c[3]);
		long long int  count = 0;
		fprintf(fptr, "\n order       No.of multiplications    ");
		mul2[0] = (r[0] * c[0] * c[1]) + (r[0] * c[1] * c[2]) + (r[0] * c[2] * c[3]);
		fprintf(fptr, "\n(((AB)C)D)\t%lld", mul2[0]);
		count++;
		fprintf(fptr, "\n");
		mul2[1] = (r[1] * c[1] * c[2]) + (r[0] * c[0] * c[2]) + (r[0] * c[2] * c[3]);
		fprintf(fptr, "\n((A(BC))D)\t%lld", mul2[1]);
		count++;
		fprintf(fptr, "\n");
		mul2[2] = (r[0] * c[0] * c[1]) + (r[2] * c[2] * c[3]) + (r[0] * c[1] * c[3]);
		fprintf(fptr, "\n((AB)(CD))\t%lld", mul2[2]);
		count++;
		fprintf(fptr, "\n");
		mul2[3] = (r[1] * c[1] * c[2]) + (r[1] * c[2] * c[3]) + (r[0] * c[0] * c[3]);
		fprintf(fptr, "\n(A((BC)D))\t%lld", mul2[3]);
		count++;
		fprintf(fptr, "\n");
		mul2[4] = (r[2] * c[2] * c[3]) + (r[1] * c[1] * c[3]) + (r[0] * c[0] * c[3]);
		fprintf(fptr, "\n(A(B(CD)))\t%lld", mul2[4]);
		count++;
		fprintf(fptr, "\n");
		int  c;
		minimum = mul2[0];

		for (c = 1; c < count; c++)
		{
			if (mul2[c] < minimum)
			{
				minimum = mul2[c];
			}
		}

		if (minimum == mul2[0])
		{
			str="(((AB)C)D)";
		}
		else if (minimum == mul2[1])
		{
			str="((A(BC))D)";
		}
		else if (minimum == mul2[2])
		{
			str="((AB)(CD))";
		}
		else if (minimum = mul2[3])
		{
			str="(A((BC)D))";
		}
		else
		{
			str="(A(B(CD)))";
		}


	}
	else if (n == 5)
	{
		fprintf(fptr, "A:%lldx%lld   B:%lldx%lld   C:%lldx%lld   D:%lldx%lld   E:%lldx%lld", r[0], c[0], r[1], c[1], r[2], c[2], r[3], c[3], r[4], c[4]);
		int  count = 0;
		fprintf(fptr, "\n order       No.of multiplications    ");
		mul3[0] = (r[2] * c[2] * c[3]) + (r[2] * c[3] * c[4]) + (r[1] * c[1] * c[4]) + (r[0] * c[0] * c[4]);
		fprintf(fptr, "\n(A(B((CD)E)))\t%lld", mul3[0]);
		count++;
		mul3[1] = (r[3] * c[3] * c[4]) + (r[2] * c[2] * c[4]) + (r[1] * c[1] * c[4]) + (r[0] * c[0] * c[4]);
		fprintf(fptr, "\n(A(B(C(DE))))\t%lld", mul3[1]);
		count++;
		mul3[2] = (r[1] * c[1] * c[2]) + (r[3] * c[3] * c[4]) + (r[1] * c[2] * c[4]) + (r[0] * c[0] * c[4]);
		fprintf(fptr, "\n(A((BC)(DE)))\t%lld", mul3[2]);
		count++;
		mul3[3] = (r[2] * c[2] * c[3]) + (r[1] * c[1] * c[3]) + (r[1] * c[3] * c[4]) + (r[0] * c[0] * c[4]);
		fprintf(fptr, "\n(A((B(CD))E))\t%lld", mul3[3]);
		count++;
		mul3[4] = (r[1] * c[1] * c[2]) + (r[1] * c[2] * c[3]) + (r[1] * c[3] * c[4]) + (r[0] * c[0] * c[4]);
		fprintf(fptr, "\n(A(((BC)D)E))\t%lld", mul3[4]);
		count++;
		mul3[5] = (r[0] * c[0] * c[1]) + (r[2] * c[2] * c[3]) + (r[2] * c[3] * c[4]) + (r[0] * c[1] * c[4]);
		fprintf(fptr, "\n((AB)((CD)E))\t%lld", mul3[5]);
		count++;
		mul3[6] = (r[0] * c[0] * c[1]) + (r[3] * c[3] * c[4]) + (r[2] * c[2] * c[4]) + (r[0] * c[1] * c[4]);
		fprintf(fptr, "\n((AB)(C(DE)))\t%lld", mul3[6]);
		count++;
		mul3[7] = (r[1] * c[1] * c[2]) + (r[0] * c[0] * c[2]) + (r[3] * c[3] * c[4]) + (r[0] * c[2] * c[4]);
		fprintf(fptr, "\n((A(BC))(DE))\t%lld", mul3[7]);
		count++;
		mul3[8] = (r[0] * c[0] * c[1]) + (r[0] * c[1] * c[2]) + (r[3] * c[3] * c[4]) + (r[0] * c[2] * c[4]);
		fprintf(fptr, "\n(((AB)C)(DE))\t%lld", mul3[8]);
		count++;
		mul3[9] = (r[0] * c[0] * c[1]) + (r[2] * c[2] * c[3]) + (r[0] * c[1] * c[3]) + (r[0] * c[3] * c[4]);
		fprintf(fptr, "\n(((AB)(CD))E)\t%lld", mul3[9]);
		count++;
		mul3[10] = (r[0] * c[0] * c[1]) + (r[0] * c[1] * c[2]) + (r[0] * c[2] * c[3]) + (r[0] * c[3] * c[4]);
		fprintf(fptr, "\n((((AB)C)D)E)\t%lld", mul3[10]);
		count++;
		mul3[11] = (r[1] * c[1] * c[2]) + (r[0] * c[0] * c[2]) + (r[0] * c[2] * c[3]) + (r[0] * c[3] * c[4]);
		fprintf(fptr, "\n(((A(BC))D)E)\t%lld", mul3[11]);
		count++;
		mul3[12] = (r[1] * c[1] * c[2]) + (r[1] * c[2] * c[3]) + (r[0] * c[0] * c[3]) + (r[0] * c[3] * c[4]);
		fprintf(fptr, "\n((A((BC)D))E)\t%lld", mul3[12]);
		count++;
		mul3[13] = (r[2] * c[2] * c[3]) + (r[1] * c[1] * c[3]) + (r[0] * c[0] * c[3]) + (r[0] * c[3] * c[4]);
		fprintf(fptr,"\n((A(B(CD)))E)\t%lld\n", mul3[13]);
		count++;
		int  c;
		minimum = mul3[0];

		for (c = 1; c < count; c++)
		{
			if (mul3[c] < minimum)
			{
				minimum = mul3[c];
			}
		}
		if (minimum == mul3[0])
		{
			str="(A(B((CD)E)))";

		}
		else if (minimum == mul3[1])
		{
			str = "(A(B(C(DE))))";
		}
		else if (minimum == mul3[2])
		{
			str = "(A((BC)(DE)))";
		}
		else if (minimum == mul3[3])
		{
			str = "(A((B(CD))E))";
		}
		else if (minimum == mul3[4])
		{
			str = "(A(((BC)D)E))";
		}
		else if (minimum == mul3[5])
		{
			str = "((AB)((CD)E))";
		}
		else if (minimum == mul3[6])
		{
			str = "((AB)(C(DE)))";
		}
		else if (minimum == mul3[7])
		{
			str = "((A(BC))(DE))";
		}
		else if (minimum == mul3[8])
		{
			str = "(((AB)C)(DE))";
		}
		else if (minimum == mul3[9])
		{
			str = "(((AB)(CD))E)";
		}
		else if (minimum == mul3[10])
		{
			str = "((((AB)C)D)E)";
		}
		else if (minimum == mul3[11])
		{
			str = "(((A(BC))D)E)";
		}
		else if (minimum == mul3[12])
		{
			str = "((A((BC)D))E)";
		}
		else if (minimum == mul3[13])
		{
			str = "((A(B(CD)))E)";
		}
	}
	fprintf(fptr, "\noptimal order: %s\noptimal cost:%lld \n", str,minimum);
	//printf("working\n");
	return str;
	
}