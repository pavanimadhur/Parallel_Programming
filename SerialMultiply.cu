
#include "hip/hip_runtime.h"


#include <stdio.h>
#include<stdlib.h>
#include <string.h>
#include <time.h>

extern void print2(long long int *mat, long long int r, long long int c, char* str);

typedef struct ll
{
	char c;
	long long int *mat;
	long long int rv;
	long long int cv;
	struct ll *next;
}node;

extern void init(int*r, int *c, long long int *a1, long long int *a2, long long int *a3, long long int *a4, long long int *a5)
{
	int i, j;
	for (i = 0; i<r[0]; i++)
	{
		for (j = 0; j<c[0]; j++)
		{
			a1[i*c[0] + j] = rand() % 2;
		}
	}
	for (i = 0; i<r[1]; i++)
	{
		for (j = 0; j<c[1]; j++)
		{
			a2[i*c[1] + j] = rand() % 3;
		}
	}
	for (i = 0; i<r[2]; i++)
	{
		for (j = 0; j<c[2]; j++)
		{
			a3[i*c[2] + j] = rand() % 4;
		}
	}
	for (i = 0; i<r[3]; i++)
	{
		for (j = 0; j<c[3]; j++)
		{
			a4[i*c[3] + j] = rand() % 5;
		}
	}
	for (i = 0; i<r[4]; i++)
	{
		for (j = 0; j<c[4]; j++)
		{
			a5[i*c[4] + j] = rand() % 6;
		}
	}
}

extern long long int* mul(long long int *mat, long long int *mat2, long long int r1, long long int c1, long long int c2, long long int *count)
{
	long long int i, j, k;
	long long int s = 0;
	long long int *res = (long long int *)malloc(r1 * c2 * sizeof(long long int));
	for (i = 0; i<r1*c2; i++)
		res[i] = 0;

	for (i = 0; i<r1; i++)
	{
		for (j = 0; j<c1; j++)
		{
			for (k = 0; k<c2; k++)
			{
				//if (mat[i*c1 + j] == 0 || mat2[j*c2 + k] == 0)
				//{
				//	s = 0;
				//	(*count)++;
				//}
				//else
				//{
					s = mat[i*c1 + j] * mat2[j*c2 + k];
				//}
				res[i*c2 + k] += s;
			}
		}
	}
	mat = (long long int *)realloc(mat, r1 * c2 * sizeof(long long int));
	for (i = 0; i<r1; i++)
	{
		for (j = 0; j<c2; j++)
		{
			mat[i*c2 + j] = res[i*c2 + j];
		}
	}
	//print2(mat, r1, c2, "program.txt");
	return mat;
}


extern void call2(int *r, int *c,char *str)
{
	//const int w=10;
	long long int i, j, final = 0, zero = 0;
	char stk[256];

	long long int *arr1 = (long long int *)malloc(r[0] * c[0] * sizeof(long long int));
	long long int *arr2 = (long long int *)malloc(r[1] * c[1] * sizeof(long long int));
	long long int *arr3 = (long long int *)malloc(r[2] * c[2] * sizeof(long long int));
	long long int *arr4 = (long long int *)malloc(r[3] * c[3] * sizeof(long long int));
	long long int *arr5 = (long long int *)malloc(r[4] * c[4] * sizeof(long long int));
	FILE *fptr;

	fptr = fopen("output.txt", "a");
	init(r, c, arr1, arr2, arr3, arr4, arr5);
	/*
	print2(arr1, r[0], c[0], "inp.txt");
	print2(arr2, r[1], c[1], "inp.txt");
	print2(arr3, r[2], c[2], "inp.txt");
	print2(arr4, r[3], c[3], "inp.txt");
	print2(arr5, r[4], c[4], "inp.txt");*/
	node *head = NULL, *temp, *temp2 = NULL, *temp3 = NULL;
	head = (node *)malloc(sizeof(node));
	head->c = 'A';
	head->mat = arr1;
	head->rv = r[0];
	head->cv = c[0];
	temp = head;
	temp->next = (node *)malloc(sizeof(node));
	temp = temp->next;
	temp->c = 'B';
	temp->rv = r[1];
	temp->cv = c[1];
	temp->mat = arr2;
	temp->next = (node *)malloc(sizeof(node));
	temp = temp->next;
	temp->c = 'C';
	temp->rv = r[2];
	temp->cv = c[2];
	temp->mat = arr3;
	temp->next = (node *)malloc(sizeof(node));
	temp = temp->next;
	temp->c = 'D';
	temp->rv = r[3];
	temp->cv = c[3];
	temp->mat = arr4;
	temp->next = NULL;
	temp->next = (node *)malloc(sizeof(node));
	temp = temp->next;
	temp->c = 'E';
	temp->rv = r[4];
	temp->cv = c[4];
	temp->mat = arr5;
	temp->next = NULL;

	//printf("enter string:\n");
	//scanf("%s", str);
	//fgets(str, sizeof(str), stdin);
	i = 0;
	j = 0;
	clock_t t;
	t = clock();
	while (i<strlen(str))
	{
		while (str[i] != ')' && i<strlen(str))
		{
			if (str[i] == '(')
			{
				i++;
			}
			else
			{
				stk[j] = str[i];
				j++;
				i++;
			}
		}
		if (str[i] == ')')
		{
			char x = stk[--j];
			char y = stk[--j];
			temp2 = head;
			temp = head;
			while (temp->next != NULL)
			{
				if (temp->c == y)
				{
					temp2 = temp;
					break;
				}
				temp = temp->next;
			}
			temp = head;
			while (temp->next != NULL)
			{
				if (temp->c == x)
				{
					temp3 = temp;
					break;
				}
				temp = temp->next;
			}
			temp2->mat = mul(temp2->mat, temp->mat, temp2->rv, temp2->cv, temp->cv, &zero);
			//final+=zero;
			temp2->cv = temp->cv;
			stk[++j] = temp2->c;
			i++;
		}
	}
	//print2(temp2->mat, temp2->rv, temp2->cv, "outp.txt");
	t = clock() - t;
	double time_taken = ((double)t) / CLOCKS_PER_SEC; // in seconds
	fprintf(fptr, "Serial Multiplication:");
	printf("Serial Multiplication:");
	fprintf(fptr,"fun() took %f seconds to execute \n", time_taken);
	printf("fun() took %f seconds to execute \n", time_taken);
	fprintf(fptr, "-------------------------------\n");
	printf("-------------------------------\n");
	//printf("number of zeroes:%lld\n", zero);
	//system("pause");
}
