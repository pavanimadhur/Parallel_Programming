#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

//#include "paranthesisation.cu"
#include <stdio.h>
#include<stdlib.h>
#include<limits.h>


extern __global__ void scalar_matrixmultiplication(int *l, long long int* Matr, int* Brack, int w,long long  int *p);
extern __global__ void MatrixMul(int *Md, int *Nd, int *Pd, int r1, int c1, int c2, int *cn);
extern int* mul(long long int *mat, long long int *mat2, long long int r1, long long int c1, long long int c2, long long int *count);
extern void call2(int *r, int *c, char *str);
extern void printParenthesis(int i, int j, int n, int bracket[50][50], char &name);
extern void print(long long int *Matrix, int len);
extern void print2(long long int *mat, long long int r, long long int c);
extern void call(int *r, int *c, char *str);
extern char* AllCombi(int n, int *r, int *c);

int compatible(int r[], int c[], int n) //to check whether the matrices can be multiplied or not
{
	int i;
	for (i = 0; i<n; i++)
	{
		if (r[i + 1] != c[i])
			return 0;
		return 1;
	}
}

int main()
{
	int n, i, count, N, r[5], c[5];
	long long int p[10], *d_p;
	int l;
	int *d_l, *d_bracket;
	int value = 2000,choice=1;
	char* str;
	long long int *d_Mat;
	hipMalloc((void **)&d_l, sizeof(int));
	hipMalloc((void **)&d_p, 10 * sizeof(long long int));

	printf("Enter no. of matrices:");
	scanf("%d", &n);

	for (i = 0; i<n; i++)
	{
		r[i]=100;
		c[i]=100;
	}
	while (value <= 35000)
	{
		if (n == 3)
		{
			if (choice == 1)
			{
				r[0] = c[1] = r[2] = value;

			}
			else if (choice == 2)
			{
				c[0] = r[1] = c[2] = value;
			}
		}
		else if (n == 4)
		{
			if (choice == 1)
			{
				r[0] = c[1] = r[2] = value;

			}
			else if (choice == 2)
			{
				c[0] = r[1] = c[2] = r[3] = value;
			}
			else if (choice == 3)
			{
				c[1] = r[2] = c[3] = value;
			}
		}
		else if (n == 5)
		{
			if (choice == 1)
			{
				r[0] = c[1] = r[2] = value;

			}
			else if (choice == 2)
			{
				c[0] = r[1] = c[2] = r[3] = value;
			}
			else if (choice == 3)
			{
				c[1] = r[2] = c[3] = r[4] = value;
			}
			else if (choice == 4)
			{
				c[2] = r[3] = c[4] = value;
			}
		}
		str = AllCombi(n, r, c);
		count = n + 1;
		int d = count*count;
		hipError_t errMemAll = hipMalloc((void **)&d_Mat, d * sizeof(long long int));
		hipError_t errMemAll2 = hipMalloc((void **)&d_bracket, d * sizeof(int));
		printf("Memory Allocation: %s\n", hipGetErrorString(errMemAll));

		long long int *Mat = (long long int *)malloc(d * sizeof(long long int));
		for (i = 0; i < d; i++)
			Mat[i] = 0;
		int *bracket = (int *)malloc(d * sizeof(int));
		for (i = 0; i < d; i++)
			bracket[i] = 0;

		for (i = 0; i < n; i++)
		{
			printf("No. of rows matrix %d:%d\n", i + 1, r[i]);
			printf("No. of columns matrix %d:%d\n", i + 1, c[i]);
		}
		if (compatible(r, c, n))
		{
			for (i = 0; i < n; i++)
				p[i] = r[i];
			p[n] = c[n - 1];
			for (l = 2; l < count; l++)//length of subchain to be multiplied
			{
				N = count - l + 1;
				hipMemcpy(d_l, &l, sizeof(int), hipMemcpyHostToDevice);
				hipMemcpy(d_p, &p, count * sizeof(long long int), hipMemcpyHostToDevice);
				hipError_t errMemDev = hipMemcpy(d_Mat, Mat, d * sizeof(long long int), hipMemcpyHostToDevice);
				hipError_t errMemDev2 = hipMemcpy(d_bracket, bracket, d * sizeof(int), hipMemcpyHostToDevice);
				printf("Memory to Device: %s\n", hipGetErrorString(errMemDev));

				scalar_matrixmultiplication << <N, 1 >> > (d_l, d_Mat, d_bracket, count, d_p);

				hipError_t errDevMem = hipMemcpy(Mat, d_Mat, d * sizeof(long long int), hipMemcpyDeviceToHost);
				hipError_t errDevMem2 = hipMemcpy(bracket, d_bracket, d * sizeof(int), hipMemcpyDeviceToHost);
				printf("Device to Memory: %s\n", hipGetErrorString(errDevMem));
			}
			long long int min = Mat[1 * count + n];
			printf("Cost of Optimal order is %lld\n", min);
			int brack2[50][50];
			for (int x = 0; x < count; x++)
				for (int y = 0; y < count; y++)
					brack2[x][y] = bracket[x*count + y];
			char name = 'A';

			printf("Optimal Parenthesization is : ");
			printParenthesis(1, count - 1, count, brack2, name);
		}

		print(Mat, count);


		free(Mat);
		free(bracket);
		hipFree(d_bracket);
		
		hipFree(d_Mat);
		
		call(r, c, str);
		//printf("stored to outp_parallel.txt\n");
		
		call2(r, c, str);
		//printf("stored to outp.txt\n");

		if (n == 3)
		{
			if (choice == 1)
			{
				r[0] = c[1] = r[2] = 100;
				choice++;
			}
			else if (choice == 2)
			{
				c[0] = r[1] = c[2] = 100;
				choice=1;
			}
		}
		else if (n == 4)
		{
			if (choice == 1)
			{
				r[0] = c[1] = r[2] = 100;
				choice++;
			}
			else if (choice == 2)
			{
				c[0] = r[1] = c[2] = r[3] = 100;
				choice++;
			}
			else if (choice == 3)
			{
				c[1] = r[2] = c[3] = 100;
				choice=1;
			}
		}
		else if (n == 5)
		{

			if (choice == 1)
			{
				r[0] = c[1] = r[2] = 100;
				choice++;
			}
			else if (choice == 2)
			{
				c[0] = r[1] = c[2] = r[3] = 100;
				choice++;
			}
			else if (choice == 3)
			{
				c[1] = r[2] = c[3] = r[4] = 100;
				choice++;
			}
			else if (choice == 4)
			{
				c[2] = r[3] = c[4] = 100;
				choice = 1;
			}
		}
		value += 3000;
	}
	hipFree(d_l);
	hipFree(d_p);
	system("pause");
	return 0;
}