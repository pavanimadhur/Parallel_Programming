#include "hip/hip_runtime.h"


#include <stdio.h>
#include<stdlib.h>
#include<limits.h>

extern __global__ void scalar_matrixmultiplication(int *l, long long int* Matr, int* Brack, int w, long long int *p)//min no of scalar multiplications
{
	int i, k, j = *l;
	long long int cost;
	i = blockIdx.x + 1;
	j = i + j - 1;
	Matr[i*w + j] = LLONG_MAX;
	for (k = i; k<j; k++)//k specifies the paranthesization value
	{
		cost = Matr[i*w + k] + Matr[(k + 1)*w + j] + p[i - 1] * p[k] * p[j];
		if (cost<Matr[i*w + j])
		{
			Matr[i*w + j] = cost;
			Brack[i*w + j] = k;
		}
	}
}


extern void printParenthesis(int i, int j, int n,
	int bracket[50][50], char &name)
{
	// If only one matrix left in current segment
	if (i == j)
	{
		printf("%c", name++);
		return;
	}

	printf("(");

	// Recursively put brackets around subexpression
	// from i to bracket[i][j].
	// Note that "*((bracket+i*n)+j)" is similar to
	// bracket[i][j]
	printParenthesis(i, bracket[i][j], n,
		bracket, name);

	// Recursively put brackets around subexpression
	// from bracket[i][j] + 1 to j.
	printParenthesis(bracket[i][j] + 1, j,
		n, bracket, name);
	printf(")");
}

extern void print(long long int *Matrix, int len)
{
	printf("\n");
	for (int i = 0; i<len*len; i++)
	{
		printf("%lld\t", Matrix[i]);
		if ((i + 1) % len == 0)
			printf("\n");
	}
}